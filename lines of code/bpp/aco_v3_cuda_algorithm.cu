#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <fstream>
#include <stdlib.h>
#include <sstream>
#include <malloc.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <ctime>
#include <chrono>

#define CUDA_ERROR_CHECK

#include "../include/aco_v3_cuda_algorithm.cuh"
#include "Randoms.cpp"

#define PHERINIT 0.005
#define EVAPORATION 0.5
#define ALPHA 1
#define BETA 1
#define TAUMAX 2
#define IROULETE 32
#define Q 100


__device__ double d_PHERINIT;
__device__ double d_EVAPORATION;
__device__ double d_ALPHA;
__device__ double d_BETA ;
__device__ double d_TAUMAX;
__device__ int d_BLOCK_SIZE;
__device__ int d_GRAPH_SIZE;

std::string::size_type sz;

int BLOCK_SIZE = 256;

Randoms *randoms;

bool is_palma = false;

using namespace std;

__global__ void setup_rand_kernel(hiprandState * state, unsigned long seed) {

	int id = blockIdx.x * blockDim.x + threadIdx.x;

	hiprand_init(seed, id, 0, &state[id]);
//	hiprand_init(1234, id, 0, &state[id]);

  if(id == 0){
	  d_PHERINIT = 0.005;
	  d_EVAPORATION = 0.5;
	  d_ALPHA = 1;
	  d_BETA = 2;
	  d_TAUMAX = 2;
  }
}

__global__ void item_copy_kernel(int* d_n_objects_types, int* d_bpp_items, int* d_bpp_items_copy){

	int ant_index = blockIdx.x * blockDim.x + threadIdx.x;
	int n_objects_type = d_n_objects_types[0];

	int bpp_items_copy_index = ant_index*n_objects_type;

	//Copy Object and Quantities array
	for(int i = 0 ; i < n_objects_type ; i++){
		int index = bpp_items_copy_index + i;
		d_bpp_items_copy[index] = d_bpp_items[i];
	}

}

// ===  FUNCTION  ======================================================================
//         Name:  packing_kernel
//         Description:
// =====================================================================================
__global__ void packing_kernel(int*	d_n_objects_types, int* d_n_objects_total,
		int* d_bins_capacity, int* d_n_ants, double* d_phero, int* d_bpp_items_weight,
		int*  d_bpp_items_quantity_copy, int*  d_bpp_items_quantity, int* d_bins, double*  d_eta, double*  d_tau, double*  d_probs,
		int* d_fitness, hiprandState* rand_states){


//	printf("\n n antes %i", d_n_ants[0]);
	int ant_index = blockIdx.x * blockDim.x + threadIdx.x;

	int n_objects_type = d_n_objects_types[0];
	int n_objects_total = d_n_objects_total[0];
	int bin_capacity = d_bins_capacity[0];

	//Actual Solution Index
	int object_bin_index = ant_index * n_objects_total; //counts the position to add new objects to the solutions where each ant has a part sized = total_objects
	int local_type_bin_index = ant_index * n_objects_type; //Counts the position of the parameters with a referent to the items = type_objects
	int bins_used = 0;

	int actual_bin_weight = 0;
	int n_items_in_actual_bin = 0;
	int remaining_capacity = 0;

	//prefix
	int bpp_items_quantity_copy_index = (int)ant_index*n_objects_type;

	//Used to check if there are still objects that could fit in the actual bin
	int possible_items_to_this_bin = 0;

	//Start first bin -> Get heaviest item available and add to first bin
	int object_index = 0;
	int object_weight = 0;
	int object_quantity = 0;
	int new_object_weight = 0;

	//Get heaviest Object to start Bin
	for(int i = 0 ; i < n_objects_type; i++){
		//Copy Object and Quantities array
		d_bpp_items_quantity_copy[bpp_items_quantity_copy_index + i] = d_bpp_items_quantity[i];

		new_object_weight = d_bpp_items_weight[i];
		//object_quantity = d_bpp_items_quantity_copy[bpp_items_quantity_copy_index + i];

		if((new_object_weight > object_weight)){
			object_index = i;
			object_weight = new_object_weight;
		}
	}

//	if(ant_index == 0){
//		printf("\n New Bin 1: \n\t Add %i - Weight %i", object_index, object_weight);
//	}

	//Add object
	d_bins[object_bin_index] = object_index;
	actual_bin_weight += object_weight;
	n_items_in_actual_bin = 1;
	bins_used = 1;

	//Remove from available itens
	d_bpp_items_quantity_copy[bpp_items_quantity_copy_index + object_index]--;

	//Loop to build complete bins
	for (int i = 1; i < n_objects_total; i++) {

		int weight_object_j = 0;
		int quantity_object_j = 0;
		int object_i = 0;

		double eta_tau_sum = 0.0;
		possible_items_to_this_bin = 0;

		remaining_capacity = (bin_capacity-actual_bin_weight);

		//Loop to check the possibility of adding other objects
		for (int index_object_j = 0; index_object_j < n_objects_type; index_object_j++) {

			//printf("\n Calc Probabilities");

			d_eta[local_type_bin_index+index_object_j] = 0.0;
			d_tau[local_type_bin_index+index_object_j] = 0.0;
			d_probs[local_type_bin_index+index_object_j] = 0.0;

			//Get data from the object list
			weight_object_j = d_bpp_items_weight[index_object_j];
			quantity_object_j = d_bpp_items_quantity_copy[bpp_items_quantity_copy_index + index_object_j];

			//Check if there is still objects available and if the weight suits actual bin
			if((quantity_object_j > 0) && (weight_object_j <= remaining_capacity)){

				//Calculate the first part of the probability calculation
				for(int k = 0 ; k < n_items_in_actual_bin ; k++){
					//last item added to by this ant = index + 1
					//Stay inside last bin using k
					object_i = d_bins[object_bin_index+i-k];

					d_eta[local_type_bin_index + index_object_j] += d_phero[object_i*n_objects_type + index_object_j];

				}
				d_eta[local_type_bin_index+index_object_j] = d_eta[local_type_bin_index+index_object_j] / n_items_in_actual_bin;
//				}

				//Calculate the second part of the probability calculation
				d_tau[local_type_bin_index + index_object_j] = (double) pow(weight_object_j, BETA);

				eta_tau_sum += d_eta[local_type_bin_index + index_object_j] * d_tau[local_type_bin_index + index_object_j];
				possible_items_to_this_bin++;
			}
		}

		if(possible_items_to_this_bin > 0){

			//Loop to Calculate probabilities based on the values calculated above
			for (int index_object_j = 0; index_object_j < n_objects_type; index_object_j++) {
				d_probs[local_type_bin_index+index_object_j] = (d_eta[local_type_bin_index+index_object_j] * d_tau[local_type_bin_index+index_object_j]) / eta_tau_sum;
			}
			//Add new object in a probabilistic manner
			double random = hiprand_uniform(&rand_states[ant_index]);
			int select_index = 0;
			int selected_object = 0;
			double sum = 0.0;
			double prob = 0.0;

			while ((sum <= random) && (select_index < n_objects_type)){

				prob = d_probs[local_type_bin_index+select_index];
				if(prob > 0.0){
					sum += prob;
					selected_object = select_index;
				}

				select_index++;
			}

			//Add selected object to the list
			d_bins[ant_index*(int)n_objects_total+i] = selected_object;

			//Add weight to actual bin
			weight_object_j = d_bpp_items_weight[selected_object];
			actual_bin_weight += weight_object_j;

//			if(ant_index == 0){
//				printf("\n\t Add %i - Weight %i", selected_object, weight_object_j);
//			}

			//Remove one available item + Increase items in Bin + reset number of possible items to the bin.
			d_bpp_items_quantity_copy[bpp_items_quantity_copy_index + selected_object]--;
			n_items_in_actual_bin++;
		}else{
//			printf("\n\n New BIN ");
			//Start new BIN
			//Start first bin -> Get heaviest item available and add to first bin
			bins_used++;

			int object_index = 0;
			int object_weight = 0;

			for(int k = 0 ; k < n_objects_type ; k++){

				object_quantity = d_bpp_items_quantity_copy[bpp_items_quantity_copy_index + k];
				weight_object_j = d_bpp_items_weight[k];

				if((object_quantity > 0) && (weight_object_j > object_weight)){
					object_index = k;
					object_weight = weight_object_j;
				}
			}

			d_bins[ant_index*(int)n_objects_total+i] = object_index;
			d_bpp_items_quantity_copy[bpp_items_quantity_copy_index + object_index]--;

			n_items_in_actual_bin = 1;
			actual_bin_weight = object_weight;

//			if(ant_index == 0){
//				printf("\n New Bin %i: \n\t Add %i - Weight %i", bins_used, object_index, object_weight);
//			}

		}
	}

	//set_fitness of current Ant
	d_fitness[ant_index] = bins_used;
}

// ===  FUNCTION  ======================================================================
//         Name:  evaporation_kernel
//         Description:
// =====================================================================================
__global__ void evaporation_kernel(double* d_phero) {

	int x_index = blockIdx.x * blockDim.x + threadIdx.x;

	//Evaporation Rate
	double RO = EVAPORATION;

	if(blockIdx.x !=  threadIdx.x){
		d_phero[x_index] = (1 - RO) * d_phero[x_index];
	}

}

// ===  FUNCTION  ======================================================================
//         Name:  evaporation_kernel
//         Description:
// =====================================================================================
__global__ void update_pheromones_kernel(int* d_n_objects_total, int* d_n_objects_type, int* d_bins_capacity,
		double* d_phero, int* d_bpp_items,int* d_bins, int* d_fitness){

	int ant_index = blockIdx.x * blockDim.x + threadIdx.x;
	int n_objects_total = d_n_objects_total[0];
	int n_objects_types = d_n_objects_type[0];
	int bin_capacity = d_bins_capacity[0];

	//double ant_fitness = d_fitness[ant_index];

	int actual_bin_weight = 0;
	int actual_bin_object_index = 0;
	int actual_bin_n_objects = 0;

	for(int i = 0 ; i < n_objects_total ; i++){
		int object_i = (int)d_bins[ant_index*n_objects_total+i];
		int object_weight = d_bpp_items[object_i];

		if(actual_bin_weight + object_weight < bin_capacity){
			actual_bin_n_objects++;
			actual_bin_weight+=object_weight;
		}else{
			//update pheromones between items from actual bin index -> n-objects
			for(int j = 0; j<actual_bin_n_objects; j++){
				for(int k = j+1; k<actual_bin_n_objects; k++){

					int object_i = d_bins[ant_index*n_objects_total+actual_bin_object_index+j];
					int object_j = d_bins[ant_index*n_objects_total+actual_bin_object_index+k];

					double delta_pheromone =  Q / d_fitness[ant_index];

					atomicAdd(&d_phero[object_i * n_objects_types + object_j],  delta_pheromone);
					atomicAdd(&d_phero[object_j * n_objects_types + object_i],  delta_pheromone);
				}
			}

			//Start new bin count
			actual_bin_n_objects = 1;
			actual_bin_weight = object_weight;
			actual_bin_object_index = i;
		}
	}
}

__global__ void update_best_fitness_kernel(int* d_n_objects_total, int* d_bins, int* d_fitness, int* d_best_fitness){

	int ant_index = blockIdx.x * blockDim.x + threadIdx.x;

	int ant_fitness = d_fitness[ant_index];

	if(ant_fitness < d_best_fitness[0]){
		d_best_fitness[0] = ant_fitness;
		//printf("\n new best %f.0", ant_fitness);
	}
}

__global__ void seq_update_best_fitness_kernel(int* d_n_ants, int* d_n_objects_total, int* d_bins, int* d_fitness, int* d_best_fitness){

	for(int i = 0 ; i < d_n_ants[0] ; i++){
		int ant_fitness = d_fitness[i];

		if(ant_fitness < d_best_fitness[0]){
			d_best_fitness[0] = ant_fitness;
		}
	}
}

// 1 step parallel reduction like in https://developer.download.nvidia.com/assets/cuda/files/reduction.pdf
// It uses shared memory
__global__ void optimized_update_best_fitness_kernel(double* d_fitness, double* d_best_fitness){
	extern __shared__ int sdata[];

	// each thread loadsone element from global to shared mem
	unsigned int tid = threadIdx.x;
	unsigned int i= blockIdx.x*blockDim.x+ threadIdx.x;

	sdata[tid] = d_fitness[i];

	__syncthreads();

	// do reduction in shared mem
	for(unsigned int s=1; s < blockDim.x; s *= 2) {
		if(tid % (2*s) == 0){
			if(sdata[tid + s] < sdata[tid]){
				sdata[tid] = sdata[tid + s];
			}
		}
		__syncthreads();
	}

	// write result for this block to global mem
	if(tid == 0){
		d_best_fitness[0] = sdata[0];
	}
}


// Starting point
int run_aco_bpp(int n_ant, int n_iterations, int problem_id, int isPalma){

	if(isPalma == 1){
		is_palma = true;
	}

	//Start GPUs
	int GPU_N;
	const int MAX_GPU_COUNT = 1;
	hipGetDeviceCount(&GPU_N);

	if (GPU_N > MAX_GPU_COUNT) {
		GPU_N = MAX_GPU_COUNT;
	}

	//printf("\n CUDA-capable device count: %i", GPU_N);
	// create stream array - create one stream per GPU
	hipStream_t stream[GPU_N];

	for (int i = 0; i < GPU_N; ++i) {
		hipSetDevice(i);
		hipDeviceReset();
		hipStreamCreate(&stream[i]);
	}

	BLOCK_SIZE = 256;

	int n_blocks = n_ant / BLOCK_SIZE;
	int n_threads = n_ant / n_blocks;

//	Create and Allocate device variables
    double* d_phero;
    double* d_delta_phero;
    double* d_probabilities;

    int* d_best_fitness;
    int* d_fitness;
    int* d_bpp_items_weight;
    int* d_bpp_items_quatity;
    int* d_bpp_items_quantity_copy;
    int* d_n_objects_type;
    int* d_n_objects_total;
    int* d_bin_capacity;

    double* d_eta;
	double* d_tau;
	double* d_sum;

    int* d_bins;
    int* d_n_ants;
    int* d_best_solution;

    randoms = new Randoms(15);

    //Initialize Host Structures and Read File in order to allocate device structures
    int* n_objects_types = (int*)malloc(1*sizeof(int));
    int* n_objects_total = (int*)malloc(1*sizeof(int));
    int* bin_capacity = (int*)malloc(1*sizeof(int));

    readBPPFileProperties(problem_id, n_objects_types, bin_capacity);

    int* bpp_items_weight = (int*)malloc(n_objects_types[0]*sizeof(int));
    int* bpp_items_quantity = (int*)malloc(n_objects_types[0]*sizeof(int));

    readBPPFile(problem_id, n_objects_types, n_objects_total, bin_capacity, bpp_items_weight, bpp_items_quantity);

    //Init Random Generators
    hiprandState* d_rand_states_ind;
	hipMalloc(&d_rand_states_ind, n_ant * sizeof(hiprandState));

	//alloc other host variables
	int bin_capacity_size = bin_capacity[0];
	int n_object_type = n_objects_types[0];
	int n_object_total = n_objects_total[0];
	int pheromone_matrix_size = n_objects_types[0] * n_objects_types[0];

	double* phero = new double[pheromone_matrix_size];
	int* best_sequence  = new int[n_object_type];

	initializePheromoneMatrix(n_object_type, phero); //Phero OK

    //alloc device variables
    hipMalloc((void**) &d_phero, pheromone_matrix_size*sizeof(double));
    hipMalloc((void**) &d_delta_phero, pheromone_matrix_size*sizeof(double));
    hipMalloc((void**) &d_fitness, n_ant*sizeof(int));
    hipMalloc((void**) &d_best_fitness, sizeof(int));
    hipMalloc((void**) &d_probabilities, n_ant * n_object_type * sizeof(double));

    hipMalloc((void**) &d_bpp_items_weight, n_object_type *sizeof(int));
    hipMalloc((void**) &d_bpp_items_quatity, n_object_type *sizeof(int));
    hipMalloc((void**) &d_bpp_items_quantity_copy, n_object_type * n_ant * sizeof(int));

	hipMalloc((void**) &d_bin_capacity, sizeof(int));
    hipMalloc((void**) &d_sum, n_ant*sizeof(double));
	hipMalloc((void**) &d_eta, n_ant*n_object_type*sizeof(double));
	hipMalloc((void**) &d_tau, n_ant*n_object_type*sizeof(double));

    hipMalloc((void**) &d_bins, n_ant* n_object_total*sizeof(int));
    hipMalloc((void**) &d_n_ants, sizeof(int));
    hipMalloc((void**) &d_n_objects_type, sizeof(int));
    hipMalloc((void**) &d_n_objects_total, sizeof(int));
    hipMalloc((void**) &d_best_solution, n_object_type*sizeof(int));

    //setup random generators TODO
    setup_rand_kernel<<<n_ant, 1, 0, stream[0]>>>(d_rand_states_ind, time(NULL));
//    checkError(0);

	int* best_fitness = (int*)malloc(1*sizeof(int));
	best_fitness[0] = 999999;

	hipMemcpy(d_best_fitness, best_fitness, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_n_objects_type, n_objects_types, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_n_objects_total, n_objects_total, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_bin_capacity, bin_capacity, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_n_ants, &n_ant, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_phero, phero, pheromone_matrix_size*sizeof(double), hipMemcpyHostToDevice);

	hipMemcpy(d_bpp_items_weight, bpp_items_weight, n_object_type*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_bpp_items_quatity, bpp_items_quantity, n_object_type*sizeof(int), hipMemcpyHostToDevice);

	hipDeviceSynchronize();

	int iteration = 0;

	double packt = 0.0;
	double evapt = 0.0;
	double updat = 0.0;
	double bestt = 0.0;

	item_copy_kernel<<<n_threads, n_blocks>>>(d_n_objects_type, d_bpp_items_quatity, d_bpp_items_quantity_copy);

	std::chrono::high_resolution_clock::time_point timer_start;
	std::chrono::high_resolution_clock::time_point timer_end;

	auto t_start = std::chrono::high_resolution_clock::now();

	//START iterations
	while(iteration < n_iterations){

		packing_kernel<<<n_threads, n_blocks>>>(d_n_objects_type, d_n_objects_total, d_bin_capacity, d_n_ants,
				d_phero, d_bpp_items_weight, d_bpp_items_quantity_copy, d_bpp_items_quatity, d_bins, d_eta, d_tau, d_probabilities,d_fitness, d_rand_states_ind);
		hipStreamSynchronize(0);


		evaporation_kernel<<<n_objects_types[0], n_objects_types[0]>>>(d_phero);

		update_pheromones_kernel<<<n_threads, n_blocks>>>(d_n_objects_total, d_n_objects_type, d_bin_capacity, d_phero, d_bpp_items_weight, d_bins, d_fitness);

		seq_update_best_fitness_kernel<<<1, 1>>>(d_n_ants, d_n_objects_total, d_bins, d_fitness, d_best_fitness);

		iteration ++;
	}

	auto t_end = std::chrono::high_resolution_clock::now();

	double time = std::chrono::duration<double>(t_end-t_start).count();
	printf(" %.4f, %.4f", packt, time);

	int bf = 0;
	hipMemcpy(&bf, d_best_fitness, sizeof(int), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

//	checkError(5);

	hipFree(d_rand_states_ind);
	hipFree(d_phero);
	hipFree(d_delta_phero);
	hipFree(d_fitness);
	hipFree(d_best_fitness);
	hipFree(d_probabilities);
	hipFree(d_bpp_items_weight);
	hipFree(d_bpp_items_quatity);
	hipFree(d_bpp_items_quantity_copy);
	hipFree(d_bin_capacity);
	hipFree(d_sum);
	hipFree(d_eta);
	hipFree(d_tau);
	hipFree(d_bins);
	hipFree(d_n_ants);
	hipFree(d_n_objects_type);
	hipFree(d_n_objects_total);
	hipFree(d_best_solution);

	checkError(6);

	free(n_objects_types);
	free(n_objects_total);
	free(bin_capacity);
	free(bpp_items_weight);
	free(bpp_items_quantity);
	free(best_fitness);

	return bf;
}
